#include "hip/hip_runtime.h"
// This file contains the GPU implementation of our op. It's a pretty typical CUDA kernel
// and I make no promises about the quality of the code or the choices made therein, but
// it should get the point accross.

#include "kepler.h"
#include "kernel_helpers.h"
#include "kernels.h"
#include "bit_helper.cu"
#include <stdio.h>

namespace kepler_jax {

namespace {

template <typename T>
__global__ void kepler_kernel(std::int64_t size, const T *mean_anom, const T *ecc, T *sin_ecc_anom,
                              T *cos_ecc_anom) {
  for (std::int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += blockDim.x * gridDim.x) {
    compute_eccentric_anomaly<T>(mean_anom[idx], ecc[idx], sin_ecc_anom + idx, cos_ecc_anom + idx);
  }
}

__device__ float cast_fp_nearest(float origin_float, int man_bits, int exp_bits,
                                 bool subnormal_support = true, bool saturate = true) {
  unsigned int target, quantize_bits;
  target = FLOAT_TO_BITS(&origin_float);
  float quantized;

  int target_exp = (target << 1 >> 1 >> 23) - 127;
  int min_exp = -((1 << (exp_bits - 1)) - 2);
  bool subnormal = (target_exp < min_exp);
  bool noquantize = (man_bits >= 23);

  if (noquantize) {
    quantized = origin_float;
  } else {
    if (subnormal && subnormal_support) {
      float shift_float, val;
      int shift_bits = ((127 + min_exp) << 23) | (target >> 31 << 31);
      shift_float = BITS_TO_FLOAT(&shift_bits);
      val = origin_float + shift_float;
      target = FLOAT_TO_BITS(&val);
      quantize_bits = round_bitwise_nearest(target, man_bits);
      quantized = BITS_TO_FLOAT(&quantize_bits) - shift_float;
    } else {
      quantize_bits = round_bitwise_nearest(target, man_bits);
      quantize_bits = clip_exponent(exp_bits, man_bits, target, quantize_bits, saturate);
      quantized = BITS_TO_FLOAT(&quantize_bits);
    }
  }

  return quantized;
}

__device__ float cast_fp_stochastic(float origin_float, unsigned int rand_prob,
                                    int man_bits, int exp_bits,
                                    bool subnormal_support = true, bool saturate = true) {
  unsigned int target, quantize_bits;
  target = FLOAT_TO_BITS(&origin_float);
  float quantized;

  int target_exp = (target << 1 >> 1 >> 23) - 127;
  int min_exp = -((1 << (exp_bits - 1)) - 2);
  bool subnormal = (target_exp < min_exp);

  if (subnormal && subnormal_support) {
    float shift_float, val;
    int shift_bits = ((127 + min_exp) << 23) | (target >> 31 << 31);
    shift_float = BITS_TO_FLOAT(&shift_bits);
    val = origin_float + shift_float;
    target = FLOAT_TO_BITS(&val);
    quantize_bits = round_bitwise_stochastic(target, rand_prob, man_bits);
    quantized = BITS_TO_FLOAT(&quantize_bits) - shift_float;
  } else {
    quantize_bits = round_bitwise_stochastic(target, rand_prob, man_bits);
    quantize_bits = clip_exponent(exp_bits, man_bits, target, quantize_bits, saturate);
    quantized = BITS_TO_FLOAT(&quantize_bits);
  }

  return quantized;
}

// quantize a float into a floating point with [exp_bits] exponent and
// [man_bits] mantissa
__global__ void float_kernel_stochastic(float *__restrict__ a,
                                        int *__restrict__ r, float *o, int size,
                                        int man_bits, int exp_bits,
                                        bool subnormal_support, bool saturate) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
    o[index] = cast_fp_stochastic(a[index], (unsigned int)r[index], man_bits,
                                  exp_bits, subnormal_support, saturate);
}

// quantize a float into a floating point with [exp_bits] exponent and
// [man_bits] mantissa
__global__ void float_kernel_nearest(const float * a, float *o, int size,
                                     int man_bits, int exp_bits,
                                     bool subnormal_support, bool saturate) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
    o[index] = cast_fp_nearest(a[index], man_bits, exp_bits, subnormal_support, saturate);
}

void ThrowIfError(hipError_t error) {
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }
}

template <typename T>
inline void apply_kepler(hipStream_t stream, void **buffers, const char *opaque,
                         std::size_t opaque_len) {
  const KeplerDescriptor &d = *UnpackDescriptor<KeplerDescriptor>(opaque, opaque_len);
  const std::int64_t size = d.size;

  const T *mean_anom = reinterpret_cast<const T *>(buffers[0]);
  const T *ecc = reinterpret_cast<const T *>(buffers[1]);
  T *sin_ecc_anom = reinterpret_cast<T *>(buffers[2]);
  T *cos_ecc_anom = reinterpret_cast<T *>(buffers[3]);

  const int block_dim = 128;
  const int grid_dim = std::min<int>(1024, (size + block_dim - 1) / block_dim);
  kepler_kernel<T>
      <<<grid_dim, block_dim, 0, stream>>>(size, mean_anom, ecc, sin_ecc_anom, cos_ecc_anom);

  ThrowIfError(hipGetLastError());
}
//Tensor a, int man_bits, int exp_bits, bool subnormals, bool saturate
inline void apply_float_quantize_nearest_cuda(hipStream_t stream, void **buffers, const char *opaque,
                         std::size_t opaque_len) {
// use external random number right now
  const KeplerDescriptor &d = *UnpackDescriptor<KeplerDescriptor>(opaque, opaque_len);
  const std::int64_t size = d.size;

  const float *a =  reinterpret_cast<const float *>(buffers[0]);
  const std::int64_t man_bits =  *reinterpret_cast<const std::int64_t *>(buffers[1]);
  const std::int64_t exp_bits =  *reinterpret_cast<const std::int64_t *>(buffers[2]);
  const bool subnormals = *reinterpret_cast<const bool *>(buffers[3]);
  const bool saturate = *reinterpret_cast<const bool *>(buffers[4]);
  float *o = reinterpret_cast<float *>(buffers[5]);

  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  float_kernel_nearest<<<blockNums, blockSize>>>(
      a, o, size, man_bits, exp_bits, 
      subnormals, saturate);
  ThrowIfError(hipGetLastError()); 
}

}  // namespace

void gpu_kepler_f32(hipStream_t stream, void **buffers, const char *opaque,
                    std::size_t opaque_len) {
  apply_kepler<float>(stream, buffers, opaque, opaque_len);
}

void gpu_kepler_f64(hipStream_t stream, void **buffers, const char *opaque,
                    std::size_t opaque_len) {
  apply_kepler<double>(stream, buffers, opaque, opaque_len);
}

 void float_quantize_nearest_cuda(hipStream_t stream, void** buffers, const char* opaque,
                    std::size_t opaque_len) {
  printf("my kernel");
  apply_float_quantize_nearest_cuda( stream, buffers, opaque, opaque_len);
}

}  // namespace kepler_jax
