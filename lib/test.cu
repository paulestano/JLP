
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    printf("hello");
    print_kernel<<<10, 10>>>();
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
    return 0;
}